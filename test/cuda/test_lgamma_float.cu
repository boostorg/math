#include "hip/hip_runtime.h"
//  Copyright John Maddock 2016.
//  Use, modification and distribution are subject to the
//  Boost Software License, Version 1.0. (See accompanying file
//  LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)

#define BOOST_MATH_OVERFLOW_ERROR_POLICY ignore_error

#include <iostream>
#include <iomanip>
#include <vector>
#include <boost/math/special_functions/gamma.hpp>
#include <boost/math/special_functions/relative_difference.hpp>
#include <boost/array.hpp>
#include "cuda_managed_ptr.hpp"
#include "stopwatch.hpp"

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

typedef float float_type;

/**
 * CUDA Kernel Device code
 *
 */
__global__ void cuda_test(const float_type *in, float_type *out, int numElements)
{
    using std::cos;
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        out[i] = boost::math::lgamma(in[i]);
    }
}

template <class T> struct table_type { typedef T type; };
typedef float_type T;
#define SC_(x) static_cast<T>(x)

#include "../test_gamma_data.ipp"

/**
 * Host main routine
 */
int main(void)
{
  try{
    // Consolidate the test data:
    std::vector<float_type> v;

    for(unsigned i = 0; i < factorials.size(); ++i)
       v.push_back(factorials[i][0]);
    for(unsigned i = 0; i < near_1.size(); ++i)
       v.push_back(near_1[i][0]);
    for(unsigned i = 0; i < near_2.size(); ++i)
       v.push_back(near_2[i][0]);
    for(unsigned i = 0; i < near_0.size(); ++i)
       v.push_back(near_0[i][0]);
    for(unsigned i = 0; i < near_m10.size(); ++i)
       v.push_back(near_m10[i][0]);
    for(unsigned i = 0; i < near_m55.size(); ++i)
       v.push_back(near_m55[i][0]);
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    std::cout << "[Vector operation on " << numElements << " elements]" << std::endl;

    // Allocate the managed input vector A
    cuda_managed_ptr<float_type> input_vector(numElements);

    // Allocate the managed output vector C
    cuda_managed_ptr<float_type> output_vector(numElements);

    // Initialize the input vectors
    for (int i = 0; i < numElements; ++i)
    {
        int table_id = i % v.size();
        input_vector[i] = v[table_id];
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 1024;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    std::cout << "CUDA kernel launch with " << blocksPerGrid << " blocks of " << threadsPerBlock << " threads" << std::endl;
    
    watch w;
    cuda_test<<<blocksPerGrid, threadsPerBlock>>>(input_vector.get(), output_vector.get(), numElements);
    hipDeviceSynchronize();
    std::cout << "CUDA kernal done in " << w.elapsed() << "s" << std::endl;
    
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "Failed to launch vectorAdd kernel (error code " << hipGetErrorString(err) << ")!" << std::endl;
        return EXIT_FAILURE;
    }

    // Verify that the result vector is correct
    std::vector<float_type> results;
    results.reserve(numElements);
    w.reset();
    for(int i = 0; i < numElements; ++i)
       results.push_back(boost::math::lgamma(input_vector[i]));
    double t = w.elapsed();
    // check the results
    for(int i = 0; i < numElements; ++i)
    {
        if (boost::math::epsilon_difference(output_vector[i], results[i]) > 300)
        {
            std::cerr << "Result verification failed at element " << i << "!" << std::endl;
            std::cerr << "Error rate was: " << boost::math::epsilon_difference(output_vector[i], results[i]) << "eps" << std::endl;
            return EXIT_FAILURE;
        }
    }

    std::cout << "Test PASSED with calculation time: " << t << "s" << std::endl;
    std::cout << "Done\n";
  }
  catch(const std::exception& e)
  {
    std::cerr << "Stopped with exception: " << e.what() << std::endl;
  }
  return 0;
}

